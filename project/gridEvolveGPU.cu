#include "hip/hip_runtime.h"
#include "gridEvolveGPU.cuh"

#define cudaCheck(expr) \
    do { \
        hipError_t e = (expr); \
        if (e != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString(e), __FILE__, __LINE__); \
            abort(); \
        } \
    } while (false)

#define BLOCKTILE  64
#define THREADTILE  4

// Given a byte representing a cell state and neighbour sum in the form:
//     <3 bits: unused><4 bits: neighbour sum><1 bit: state>
// this value maps the byte to 1 if the state will change, else 0
#define TRANSITION_MAP 0x2AA4A  // bin: 101010101001001010


__host__ void gridEvolve_GPU(char *grid, char *temp, const int nRows, const int nCols, char *isStatic)
{
    // NOTE: main script handles all hipMemcpy'ing (before, during and after)

    // // Initially set isStatic as true so any thread can set it false
    // *isStatic = 1;

    // NOTE: nCols and nRows must both be a multiple of BLOCKTILE
    dim3 numBlocks(nCols / BLOCKTILE, nRows / BLOCKTILE);
    dim3 threadsPerBlock(BLOCKTILE * BLOCKTILE / (THREADTILE * THREADTILE));
    gridEvolveKernel_GPU<<<numBlocks, threadsPerBlock>>>(grid, temp, nRows, nCols, isStatic);
}

__global__ void gridEvolveKernel_GPU(char *grid, char *temp, const int nRows, const int nCols, char *isStatic)
{
    const int rowSize = nCols + 2;

    // Indices of current thread within its block
    const int threadRow = threadIdx.x / (BLOCKTILE / THREADTILE);
    const int threadCol = threadIdx.x % (BLOCKTILE / THREADTILE);

    // Move pointers to grids to relevant positions for this thread block,
    // (... + rowSize + 1) to account for zero-padding
    const int blockStartIdx = (blockIdx.y * BLOCKTILE) * rowSize +
                              (blockIdx.x * BLOCKTILE) + rowSize + 1;
    grid += blockStartIdx;
    temp += blockStartIdx;

    // // Define local equivalent to avoid unnecessary global memory accesses
    // char locallyStatic = 1;

    // Thread to address cells that it is responsible for
    for (int row = 0; row < THREADTILE; row++)
    {
        for (int col = 0; col < THREADTILE; col++)
        {
            const int idx = (threadRow * THREADTILE + row) * rowSize +
                             threadCol * THREADTILE + col;
            const int state = grid[idx];

            // Determine neighbourhood sum
            const int idx_abv = idx - rowSize;
            const int idx_blw = idx + rowSize;
            char neighbourSum = 0;
            neighbourSum += grid[idx_abv - 1];
            neighbourSum += grid[idx_abv];
            neighbourSum += grid[idx_abv + 1];
            neighbourSum += grid[idx - 1];
            neighbourSum += grid[idx + 1];
            neighbourSum += grid[idx_blw - 1];
            neighbourSum += grid[idx_blw];
            neighbourSum += grid[idx_blw + 1];

            // TODO: optimise by loading all cells that need to be checked
            //  (i.e. 6x6 bytes) into cache. Can also go a step further and load
            //  all cells needed by a block into shared memory. Actually, might
            //  be better trying this vice versa. This might also lend itself
            //  well to not needing a temp grid at all.

            // Lookup if state changes
            if (TRANSITION_MAP & (1 << ((neighbourSum << 1) | state)))
            {
                temp[idx] = !state;
                // locallyStatic = 0;
            }
        }
    }

    // // Update grid static status (multiple threads trying to write doesn't
    // // matter as long as at least one manages to write)
    // if (!locallyStatic)
    // {
    //     *isStatic = 0;
    // }
}
